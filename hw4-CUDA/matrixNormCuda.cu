/* Matrix normalization.
* Compile with "nvcc matrixNormCuda.c -lm"
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <math.h>

/* Program Parameters */
#define N 8000  /* Matrix size */
int blocks_per_grid = 32;
int threads_per_block = 256; 

/* Matrices */
float A[N*N], B[N*N];

/* CUDA arrays */
float *A_d, *B_d;


/* Initialize A and B*/
void initialize_inputs() {
    int row, col;
    
    // srand((unsigned)time(NULL));
    for (row = 0; row < N; row++) {
        for (col = 0; col < N; col++) {
            A[row*N + col] = (float)rand() / 32768.0;
            B[row*N + col] = 0.0;
        }
    }
    
}

/* Print input matrices */
void print_inputs()
{
  int row, col;
  int howmuchtoprint = 10;
    printf("\nA =\n\t");
    for (row = 0; row < howmuchtoprint; row++)
    {
      for (col = 0; col < howmuchtoprint; col++)
      {
        printf("%5.2f%s", A[row*N + col], (col < howmuchtoprint - 1) ? ", " : ";\n\t");
      }
    }
    printf("\nB = [");
    for (row = 0; row < howmuchtoprint; row ++)
    {
        for (col = 0; col < howmuchtoprint; col++)
        {
        printf("%5.2f%s", B[row*N + col], (col < howmuchtoprint - 1 ) ? "; " : "]\n");
        }
    }

}

/* Kernel function */

__global__ void matrixNorm(float* A_dd, float* B_dd, int N_d) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // __shared__ float mu, sigma;
    float mu, sigma;
    int row;

    if (idx < N_d) {
        mu = 0.0;
        for (row=0; row < N_d; row++){
            mu += A_dd[row*N_d + idx];
        }
        mu /= N_d;

        sigma = 0.0;
        for (row=0; row < N_d; row++){
            sigma += powf(A_dd[row*N_d + idx] - mu, 2.0);
        }
        sigma /= N_d;
        sigma = sqrt(sigma);

        for (row=0; row < N_d; row++) {
            if (sigma == 0.0){
                B_dd[row*N_d + idx] = 0.0;
            }
            else{
                B_dd[row*N_d + idx] = (A_dd[row*N_d + idx] - mu) / sigma;
            }
        }
    }
}



int main(int argc, char **argv) {
    /* Timing variables */
    struct timeval start, stop;  /* Elapsed times using gettimeofday() */
    struct timezone tzdummy;
    unsigned long long runtime;
    // int col, row;
    // float mu, sigma;
    
    /* Initialize A and B */
    initialize_inputs();
    print_inputs();
    
    /* Start Clock */
    printf("\n---------------------------------------------\n");
    printf("Matrix size N = %d", N);
    printf("\nStarting clock.\n\n");
    gettimeofday(&start, &tzdummy);
    
    printf("Computing Parallely.\n");
    
    /*allocating GPU space*/
    hipMalloc((void **) &A_d, N*N*sizeof(float));
    hipMalloc((void **) &B_d, N*N*sizeof(float));

    /*transfer data from host to device*/
    hipMemcpy(A_d,A,N*N*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(B_d,B,N*N*sizeof(float),hipMemcpyHostToDevice);

    /* Kernal Matrix Normalization */
    matrixNorm<<<blocks_per_grid,threads_per_block>>>(A_d,B_d,N);

    //note to self: KERNAL CALLS ARE EXPENSIVE AF
    // for (col=0; col < N; col++) {
    //     mu = 0.0;
    //     for (row=0; row < N; row++)
    //         mu += A[row][col];
    //     mu /= (float) N;
    //     sigma = 0.0;
    //     for (row=0; row < N; row++)
    //         sigma += powf(A[row][col] - mu, 2.0);
    //     sigma /= (float) N;
    //     sigma = sqrt(sigma);
    //     matrixNorm<<<blocks_per_grid,threads_per_block>>>(A_d,B_d,mu,sigma, N);
    // }

    /*transfer data from device to host*/
    hipMemcpy(B,B_d,N*N*sizeof(float),hipMemcpyDeviceToHost);
    // cudaMemcpy(A,A_d,N*N*sizeof(float),cudaMemcpyDeviceToHost);
    
    /*deallocating GPU space*/
    hipFree(A_d);
    hipFree(B_d);
    
    printf("\n-------Output--------------------------------------------\n");
    print_inputs();
    
    /* Stop Clock */
    gettimeofday(&stop, &tzdummy);
    runtime = (unsigned long long)(stop.tv_sec - start.tv_sec) * 1000000 + (stop.tv_usec - start.tv_usec);
    
    
    /* Display timing results */
    printf("Runtime = %g ms.\n", (float)runtime/(float)1000);
    printf("\nStopped clock.");
    printf("\n---------------------------------------------\n");
    
    exit(0);
}














